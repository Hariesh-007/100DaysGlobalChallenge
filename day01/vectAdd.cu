#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vecAdd(float* a, float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int n = 10;
    float a[10] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    float b[10] = {10, 9, 8, 7, 6, 5, 4, 3, 2, 1};
    float c[10];
    
    float *dev_a, *dev_b, *dev_c;
    
    hipMalloc(&dev_a, n * sizeof(float));
    hipMalloc(&dev_b, n * sizeof(float)); 
    hipMalloc(&dev_c, n * sizeof(float));
    
    hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice);
    
    int block_size = 32;
    int grid_size = (n + block_size - 1) / block_size;
    vecAdd<<<grid_size, block_size>>>(dev_a, dev_b, dev_c, n);
    
    hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost);
    
    std::cout << "First few results: ";
    for(int i = 0; i < 5; i++) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;
    
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    return 0;
}
